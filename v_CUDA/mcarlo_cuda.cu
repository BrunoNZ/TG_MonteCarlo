#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "/opt/NVIDIA_GPU_Computing_SDK/C/src/simplePrintf/cuPrintf.cu"
#include "declaracoes.h"

/*------------------------------------------------------------------*/
/*	FUNCOES AUXILIARES												*/
/*------------------------------------------------------------------*/
int calcula_pos_matriz(int NT, int p, int t){
	return (p*NT)+t;
}

void le_argumentos(int argc, char **argv, parametros *param) {
	if (argc-1 != 8){
		printf("PARAMETROS INVALIDOS!\n");
		printf("Use: %s <ARQ_SERIE_ENTRADA> <ARQ_MATRIZ_ENTRADA> <ARQ_SAIDA> <NX> <NY> <NT> <UNDEF> <OP_TAM>\n",argv[0]);
		exit(1);
	}
	
	param->arq_serie_entrada=argv[1];
	param->arq_matriz_entrada=argv[2];
	param->arq_saida=argv[3];
	param->NP=atoi(argv[4])*atoi(argv[5]); //NP = NX * NY
	param->NT=atoi(argv[6]);
	param->UNDEF=atof(argv[7]);
	
	switch (atoi(argv[8])){  //argv[8] = OP_TAM 
		case 15 : param->TAM_MAX=(size_t)_15MB; break;
		case 25 : param->TAM_MAX=(size_t)_25MB; break;
		case 50 : param->TAM_MAX=(size_t)_50MB; break;
		case 100 : param->TAM_MAX=(size_t)_100MB; break;
		case 150 : param->TAM_MAX=(size_t)_150MB; break;
		case 200 : param->TAM_MAX=(size_t)_200MB; break;
		case 250 : param->TAM_MAX=(size_t)_250MB; break;
		default : printf("ERRO! Opcao de tamanho maximo invalida!\n"); exit(1);
	}
}

void le_serie_entrada(parametros param, float **s){
	FILE *arq;
	
	arq=fopen(param.arq_serie_entrada,"rb");
	if (!arq){
		printf("Erro na abertura do arquivo da serie de entrada : \"%s\".\n",param.arq_serie_entrada);
		exit (1);
	}
	
	(*s)=(float*)malloc((param.NT)*sizeof(float));
	
	fread((*s),sizeof(float),param.NT,arq);
	
	fclose(arq);
}

void le_matriz_entrada(parametros param, float **d){
	int p,t,pos;
	FILE *arq;
	float *buffer;

	arq=fopen(param.arq_matriz_entrada,"rb");
	if (!arq){
		printf("Erro na abertura do arquivo da serie de entrada : \"%s\".\n",param.arq_matriz_entrada);
		exit (1);
	}
	
	buffer=(float*)malloc(param.NP*sizeof(float));

	(*d)=(float*)malloc((param.NP*param.NT)*sizeof(float));
	
	/*
	O arquivo binario esta organizado da seguinte maneira:
	[1,1,1][2,1,1][3,1,1],...,[NX,1,1]
	[1,2,1][2,2,1][3,2,1],...,[NX,2,1]
	[1,3,1][2,3,1][3,3,1],...,[NX,3,1]
	...
	[1,NY,1][2,NY,1][3,NY,1],...,[NX,NY,1]
	[1,1,2][2,1,2][3,1,2],...,[NX,1,2]
	[1,2,2][2,2,2][3,2,2],...,[NX,2,2]
	[1,3,2][2,3,2][3,3,2],...,[NX,3,2]
	...
	[1,NY,2][2,NY,2][3,NY,2],...,[NX,NY,2]
	[1,1,3][2,1,3][3,1,3],...,[NX,1,3]
	[1,2,3][2,2,3][3,2,3],...,[NX,2,3]
	[1,3,3][2,3,3][3,3,3],...,[NX,3,3]
	...
	[1,NY,2][2,NY,2][3,NY,NT-2],...,[NX,NY,NT-1]
	[1,1,3][2,1,3][3,1,NT-1],...,[NX,1,NT-1]
	[1,2,3][2,2,3][3,2,NT-1],...,[NX,2,NT-1]
	[1,3,3][2,3,3][3,3,NT-1],...,[NX,3,NT-1]
	...
	[1,NY,NT][2,NY,NT][3,NY,NT],...,[NX,NY,NT]

	Ou seja, varia varia primeiro o X, depois o Y, e por ultimo o T.
	*/
		
	for (t=0;t<param.NT;t++){
		fread(buffer,sizeof(float),param.NP,arq);

		for(p=0;p<param.NP;p++){
			pos=calcula_pos_matriz(param.NT,p,t);
			(*d)[pos]=buffer[p];
		}
	}

	free(buffer);
	
	fclose(arq);
}

void salva_arq_saida(parametros param, float *s){
	FILE *arq;
		
	arq=fopen(param.arq_saida,"wb");
	if (!arq){
		printf("Erro na abertura do arquivo de saida.\n");
		exit (1);
	}

	fwrite(s,sizeof(float),param.NP,arq);
		
	fclose(arq);	
}

void desaloca_variaveis(){
	if (h_serie_entrada != NULL) free(h_serie_entrada);
	if (h_matriz_entrada != NULL) free(h_matriz_entrada);
	if (h_saida != NULL) free(h_saida);
	
	if (d_serie_entrada != NULL) hipFree(d_serie_entrada);
	if (d_matriz_entrada != NULL) hipFree(d_matriz_entrada);
	if (d_saida != NULL) hipFree(d_saida);
}

/*------------------------------------------------------------------*/
/*------------------------------------------------------------------*/

/*------------------------------------------------------------------*/
/*	FUNCOES PARA DEBUGAR											*/
/*------------------------------------------------------------------*/
void imprime_dados_host(float *d, int p, int NT_IMPRESSAO, int NT){
	int t, ini_seq;
	ini_seq=(p*NT);
	for (t=0;t<NT_IMPRESSAO;t++)
		printf("[HOST] DADOS[%d][%d] (POS=%d) = %f\n",p,t,ini_seq+t,d[ini_seq+t]);

}
/*
__global__ void imprime_dados_device(const float *d, int p, int NT_IMPRESSAO, int NT){
	int t, ini_seq;
	ini_seq=(p*NT);
	for (t=200;t<NT_IMPRESSAO;t++)
		cuPrintf("[DEVICE] DADOS[%d][%d] (POS=%d) = %f\n",p,t,ini_seq+t,d[ini_seq+t]);
	
}
*/

/*
__global__ void imprime_dimensoes_execucao(){
	int p = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (blockIdx.x == 0 && threadIdx.x == 0)
		//cuPrintf("blockDim: %d / blockIdx: %d / threadIdx: %d / P: %d\n",
					blockDim.x,blockIdx.x,threadIdx.x,p);
}
*/

void imprime_argumentos(parametros param){
	printf("\n");
	printf("ARGUMENTOS:\n");
	printf("-> ARQ_SERIE_ENTRADA: %s\n",param.arq_serie_entrada);
	printf("-> ARQ_MATRIZ_ENTRADA: %s\n",param.arq_matriz_entrada);
	printf("-> ARQ_SAIDA: %s\n",param.arq_saida);
	printf("-> DIMENSOES: %d x %d\n",param.NP,param.NT);
	printf("-> UNDEF: %f\n",param.UNDEF);
	printf("\n");
}

void imprime_parametros_execucao(parametros_exec p_exec){
	printf("PARAMETROS DE EXECUCAO:\n");
    printf(" > TOTAL_POS = %d\n",p_exec.npos_por_ciclo);
    printf(" > TOTAL_CICLOS = %d\n",p_exec.total_ciclos);
    printf(" > THREADS_POR_BLOCO = %d\n",p_exec.threads_por_bloco);
    printf(" > BLOCOS_POR GRID = %d\n",p_exec.blocos_por_grid);
    printf(" > TAM_POR_CICLO = %ld\n",p_exec.tam_por_ciclo);
    printf("\n");
}
/*------------------------------------------------------------------*/
/*------------------------------------------------------------------*/

/*------------------------------------------------------------------*/
/*	FUNCOES AUXILIARES ESPECIFICAS DO CUDA							*/
/*------------------------------------------------------------------*/
void verifica_erro_cuda(char *F_ID, int ciclo){
		hipError_t erro = hipGetLastError();
		if(hipSuccess != erro){
			printf( "[CUDA_ERROR]: %s\n", hipGetErrorString(erro) );
			printf( "ID: %s (CICLO: %d)\n", F_ID, ciclo);
			desaloca_variaveis();
			exit(1);
		}
}

void calcula_parametros_execucao(parametros param, parametros_exec *param_exec){
									
	int npos_por_ciclo;
	int total_ciclos;
	size_t tam_por_ciclo;
	int threads_por_bloco;
	int blocos_por_grid;
	
	total_ciclos=1;
	npos_por_ciclo=param.NP;

	tam_por_ciclo=(npos_por_ciclo*param.NT)*sizeof(float);
	while ( tam_por_ciclo > param.TAM_MAX ){
		total_ciclos=total_ciclos+1;
		npos_por_ciclo=ceil(param.NP/total_ciclos);
		tam_por_ciclo=(npos_por_ciclo*param.NT)*sizeof(float);
	}
		
    threads_por_bloco = 8;
    blocos_por_grid = (npos_por_ciclo+threads_por_bloco-1)/threads_por_bloco;
           
    param_exec->npos_por_ciclo=npos_por_ciclo;
	param_exec->total_ciclos=total_ciclos;
	param_exec->tam_por_ciclo=tam_por_ciclo;
	param_exec->threads_por_bloco=threads_por_bloco;
	param_exec->blocos_por_grid=blocos_por_grid;

	return;
}
/*------------------------------------------------------------------*/
/*------------------------------------------------------------------*/

/*------------------------------------------------------------------*/
/* FUNCOES PARA O CALCULO DA CORRELACAO E SIGNIFICANCIA				*/
/*------------------------------------------------------------------*/

/*
__device__ void shuffle(float *array, size_t n){
    if (n > 1) {
        size_t i;
		for (i = 0; i < n - 1; i++) {
			size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
			float t = array[j];
			array[j] = array[i];
			array[i] = t;
		}
    }
}
*/

__device__ float correlacao_serie_serie(float *vA, float *vB, int nt, float undef){
	
	int i,k;
	float E_cov, E_anm2_A, E_anm2_B, div;
	float med_A, med_B;
	
	//CRIA DOIS VETORES AUXILIARES APENAS COM AS "DUPLAS" DE
	//DADOS ONDE AMBOS OS VALORES NAO SEJAM INDEFINIDOS.
	//ALEM DISSO JA FAZ O SOMATORIO DAS SERIES FINAIS PARA DEPOIS
	//CALCULAR A MEDIA;
	//*** k = Numero de duplas validas e tamanho total das series auxiliares ***
	med_A=0.0;
	med_B=0.0;
	k=0;
	for (i=0;i<nt;i++){
		if ((vA[i] != undef) && (vB[i] != undef)){
			med_A=med_A+(float)vA[i];
			med_B=med_B+(float)vB[i];
					
			k++;
		}
	}
	if (k < 2)
		return (float)undef;
		
	med_A=med_A/(float)k;
	med_B=med_B/(float)k;
	E_cov=0.0;
	E_anm2_A=0.0;
	E_anm2_B=0.0;
	for (i=0;i<k;i++){
		if ((vA[i] != undef) && (vB[i] != undef)){
			E_cov    = E_cov	+ (((float)vA[i]-med_A)*((float)vB[i]-med_B));
			E_anm2_A = E_anm2_A	+ (((float)vA[i]-med_A)*((float)vA[i]-med_A));
			E_anm2_B = E_anm2_B	+ (((float)vB[i]-med_B)*((float)vB[i]-med_B));
		}
	}
	
	div=sqrt(E_anm2_A * E_anm2_B);
	if (div == 0) return undef;
	else return (float)(E_cov/div);
}

__global__ void sig_mcarlo_serie_serie(
					float *vA, float *vB, int nt, float undef, int total_perm,
					int total_pos, float *saida){
	
	int i, cont;
	float correl_orig, correl;
	
	int p = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (p >= total_pos) return;
	
	correl_orig=correlacao_serie_serie(vA,vB,nt,undef);

	if (correl_orig == undef) saida[p]=undef;
	
	/*
	//srand(time(NULL));
	for (i=0;i<total_perm;i++){
		//shuffle(vB,nt);
		correl=correlacao_serie_serie(vA,vB,nt,undef);
		if (fabs(correl) >= fabs(correl_orig)) cont++;
	}
	*/
	
	//saida[p]=((float)cont/(float)total_perm)*(correl_orig*fabs(correl_orig));
	saida[p]=(float)p;
}

/*------------------------------------------------------------------*/
/*------------------------------------------------------------------*/

int main(int argc, char **argv){
	
	int pos_i, total_npos, ciclo;
	int pos_inicio_copia_entrada, pos_inicio_copia_saida;

	//LE OS ARGUMENTOS E OS DADOS DE ENTRADA
	le_argumentos(argc,argv,&param);
	imprime_argumentos(param);
	
	le_serie_entrada(param,&h_serie_entrada);
	le_matriz_entrada(param,&h_matriz_entrada);
	
	//CALCULA OS PARAMETROS DE EXECUCAO
	calcula_parametros_execucao(param, &param_exec);
	imprime_parametros_execucao(param_exec);
	
	//CALCULA O NUMERO TOTAL DE POSICOES QUE SERAO CALCULADAS
	// *** ESSE NUMERO PROVAVELMENTE SERA MAIOR QUE O NP
	total_npos=param_exec.npos_por_ciclo*param_exec.total_ciclos;
	
	//ALOCA O ESPACO PARA OS DADOS DE ENTRADA E SAIDA NO DEVICE
	hipMalloc((void**)&d_serie_entrada, param.NT*sizeof(float) );
		verifica_erro_cuda("hipMalloc(d_serie_entrada)",-1);
	hipMalloc((void**)&d_matriz_entrada, param_exec.tam_por_ciclo );
		verifica_erro_cuda("hipMalloc(d_matriz_entrada)",-1);
	hipMalloc((void**)&d_saida, total_npos*sizeof(float) );
		verifica_erro_cuda("hipMalloc(d_saida)",-1);
	
	//COPIA OS DADOS DA SERIE DE ENTRADA
	hipMemcpy(d_serie_entrada,h_serie_entrada,
		param.NT*sizeof(float), hipMemcpyHostToDevice);
		verifica_erro_cuda("hipMemcpy(d_serie_entrada)",-1);
	
	//ALOCA NO HOST O ESPACO PARA A SAIDA (TOTAL)
	h_saida=(float*)malloc(param.NP*sizeof(float));

	pos_i=0;
	for (ciclo=0;ciclo<param_exec.total_ciclos;ciclo++){
		
		//RESETA A MEMORIA DO DEVICE
		hipMemset(d_matriz_entrada,param.UNDEF,param_exec.tam_por_ciclo);
			verifica_erro_cuda("hipMemset(d_matriz_entrada)",ciclo);
		hipMemset(d_saida,param.UNDEF,param_exec.npos_por_ciclo*sizeof(float));
			verifica_erro_cuda("hipMemset(d_saida)",ciclo);

		//CALCULA A POSICAO DE INICIO DA COPIA DOS DADOS DE ENTRADA
		pos_inicio_copia_entrada=pos_i*param.NT;
		
		//COPIA OS DADOS DA MATRIZ DE ENTRADA
		hipMemcpy(d_matriz_entrada,(h_matriz_entrada+pos_inicio_copia_entrada),
			param_exec.tam_por_ciclo, hipMemcpyHostToDevice);
			verifica_erro_cuda("hipMemcpy(d_matriz_entrada)",ciclo);

		//EXECUTA O MCARLO EM CUDA
		/*
		sig_mcarlo_serie_serie<<<param_exec.blocos_por_grid, param_exec.threads_por_bloco>>>(
			h_serie_entrada, h_matriz_entrada, param.NT, param.UNDEF, TOTAL_PERM,
			param_exec.npos_por_ciclo, d_saida);
			verifica_erro_cuda("sig_mcarlo_serie_serie",ciclo);
		*/
			
		//ESPERA O TERMINIO DA EXECUCAO DO DEVICE
		hipDeviceSynchronize();
		
		//CALCULA A POSICAO DE INICIO DA COPIA DOS DADOS DE SAIDA
		pos_inicio_copia_saida=ciclo*param_exec.npos_por_ciclo;

		//COPIA A SAIDA DO DEVICE PARA O HOST
		hipMemcpy((h_saida+pos_inicio_copia_saida),d_saida,
			param_exec.npos_por_ciclo*sizeof(float), hipMemcpyDeviceToHost);
			verifica_erro_cuda("hipMemcpy(h_saida)",ciclo);
		
		//CALCULA A POSICAO INICIAL PARA O PROXIMO CICLO
		pos_i=pos_i+param_exec.npos_por_ciclo;
	}
	
	//for (ciclo=0;ciclo<param.NP;ciclo++)
	//	printf("%d -> %f\n",ciclo,h_saida[ciclo]);
	
	//salva_arq_saida(param, h_saida);
	
	desaloca_variaveis();
	
	printf("FIM!\n");
		
	return 1;
}
